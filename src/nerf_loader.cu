#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   nerfloader.cu
 *  @author Alex Evans & Thomas Müller, NVIDIA
 *  @brief  Loads a NeRF data set from NeRF's original format
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/thread_pool.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>

#include <json/json.hpp>

#include <filesystem/path.h>

#define _USE_MATH_DEFINES
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#define STB_IMAGE_IMPLEMENTATION

#if defined(__NVCC__)
#if defined __NVCC_DIAG_PRAGMA_SUPPORT__
#  pragma nv_diag_suppress 550
#else
#  pragma diag_suppress 550
#endif
#endif
#include <stb_image/stb_image.h>
#if defined(__NVCC__)
#if defined __NVCC_DIAG_PRAGMA_SUPPORT__
#  pragma nv_diag_default 550
#else
#  pragma diag_default 550
#endif
#endif

using namespace tcnn;
using namespace std::literals;
using namespace Eigen;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

__global__ void convert_rgba32(const uint64_t num_pixels, const uint8_t* __restrict__ pixels, uint8_t* __restrict__ out, bool white_2_transparent = false, bool black_2_transparent = false, uint32_t mask_color = 0) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_pixels) return;

	uint8_t rgba[4];
	*((uint32_t*)&rgba[0]) = *((uint32_t*)&pixels[i*4]);

	// NSVF dataset has 'white = transparent' madness
	if (white_2_transparent && rgba[0] == 255 && rgba[1] == 255 && rgba[2] == 255) {
		rgba[3] = 0;
	}

	if (black_2_transparent && rgba[0] == 0 && rgba[1] == 0 && rgba[2] == 0) {
		rgba[3] = 0;
	}

	if (mask_color != 0 && mask_color == *((uint32_t*)&rgba[0])) {
		// turn the mask into hot pink
		rgba[0] = 0xFF; rgba[1] = 0x00; rgba[2] = 0xFF; rgba[3] = 0x00;
	}

	*((uint32_t*)&out[i*4]) = *((uint32_t*)&rgba[0]);
}

__global__ void convert_rgba64(const uint64_t num_pixels, const uint16_t* __restrict__ pixels, uint16_t* __restrict__ out, bool white_2_transparent = false, bool black_2_transparent = false, uint64_t mask_color = 0) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_pixels) return;

	uint16_t rgba[4];
	*((uint64_t*)&rgba[0]) = *((uint64_t*)&pixels[i*4]);

	// NSVF dataset has 'white = transparent' madness
	if (white_2_transparent && rgba[0] == 65535 && rgba[1] == 65535 && rgba[2] == 65535) {
		rgba[3] = 0;
	}

	if (black_2_transparent && rgba[0] == 0 && rgba[1] == 0 && rgba[2] == 0) {
		rgba[3] = 0;
	}

	if (mask_color != 0 && mask_color == *((uint64_t*)&rgba[0])) {
		// turn the mask into hot pink
		rgba[0] = 0xFF; rgba[1] = 0x00; rgba[2] = 0xFF; rgba[3] = 0x00;
	}

	*((uint64_t*)&out[i*4]) = *((uint64_t*)&rgba[0]);
}

__global__ void from_fullp(const uint64_t num_elements, const float* __restrict__ pixels, __half* __restrict__ out) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	out[i] = (__half)pixels[i];
}

template <typename T>
__global__ void copy_depth(const uint64_t num_elements, float* __restrict__ depth_dst, const T* __restrict__ depth_pixels, float depth_scale) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	if (depth_pixels == nullptr || depth_scale <= 0.f) {
		depth_dst[i] = 0.f; // no depth data for this entire image. zero it out
	} else {
		depth_dst[i] = depth_pixels[i] * depth_scale;
	}
}

template <typename T>
__global__ void sharpen(const uint64_t num_pixels, const uint32_t w, const T* __restrict__ pix, T* __restrict__ destpix, float center_w, float inv_totalw) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_pixels) return;

	float rgba[4] = {
		(float)pix[i*4+0]*center_w,
		(float)pix[i*4+1]*center_w,
		(float)pix[i*4+2]*center_w,
		(float)pix[i*4+3]*center_w
	};

	int64_t i2=i-1; if (i2<0) i2=0; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	i2=i-w; if (i2<0) i2=0; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	i2=i+1; if (i2>=num_pixels) i2-=num_pixels; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	i2=i+w; if (i2>=num_pixels) i2-=num_pixels; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	for (int j=0;j<4;++j) destpix[i*4+j]=(T)max(0.f, rgba[j] * inv_totalw);
}

__device__ inline float luma(const Array4f& c) {
	return c[0] * 0.2126f + c[1] * 0.7152f + c[2] * 0.0722f;
}

__global__ void compute_sharpness(Eigen::Vector2i sharpness_resolution, Eigen::Vector2i image_resolution, uint32_t n_images, const void* __restrict__ images_data, EImageDataType image_data_type, float* __restrict__ sharpness_data) {
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t i = threadIdx.z + blockIdx.z * blockDim.z;
	if (x >= sharpness_resolution.x() || y >= sharpness_resolution.y() || i>=n_images) return;
	const size_t sharp_size = sharpness_resolution.x() * sharpness_resolution.y();
	sharpness_data += sharp_size * i + x + y * sharpness_resolution.x();

	// overlap patches a bit
	int x_border = 0; // (image_resolution.x()/sharpness_resolution.x())/4;
	int y_border = 0; // (image_resolution.y()/sharpness_resolution.y())/4;

	int x1 = (x*image_resolution.x())/sharpness_resolution.x()-x_border, x2 = ((x+1)*image_resolution.x())/sharpness_resolution.x()+x_border;
	int y1 = (y*image_resolution.y())/sharpness_resolution.y()-y_border, y2 = ((y+1)*image_resolution.y())/sharpness_resolution.y()+y_border;
	// clamp to 1 pixel in from edge
	x1=max(x1,1); y1=max(y1,1);
	x2=min(x2,image_resolution.x()-2); y2=min(y2,image_resolution.y()-2);
	// yes, yes I know I should do a parallel reduction and shared memory and stuff. but we have so many tiles in flight, and this is load-time, meh.
	float tot_lap=0.f,tot_lap2=0.f,tot_lum=0.f;
	float scal=1.f/((x2-x1)*(y2-y1));
	for (int yy=y1;yy<y2;++yy) {
		for (int xx=x1; xx<x2; ++xx) {
			Array4f n, e, s, w, c;
			c = read_rgba(Vector2i{xx, yy}, image_resolution, images_data, image_data_type, i);
			n = read_rgba(Vector2i{xx, yy-1}, image_resolution, images_data, image_data_type, i);
			w = read_rgba(Vector2i{xx-1, yy}, image_resolution, images_data, image_data_type, i);
			s = read_rgba(Vector2i{xx, yy+1}, image_resolution, images_data, image_data_type, i);
			e = read_rgba(Vector2i{xx+1, yy}, image_resolution, images_data, image_data_type, i);
			float lum = luma(c);
			float lap = lum * 4.f - luma(n) - luma(e) - luma(s) - luma(w);
			tot_lap += lap;
			tot_lap2 += lap*lap;
			tot_lum += lum;
		}
	}
	tot_lap*=scal;
	tot_lap2*=scal;
	tot_lum*=scal;
	float variance_of_laplacian = tot_lap2 - tot_lap * tot_lap;
	*sharpness_data = (variance_of_laplacian) ; // / max(0.00001f,tot_lum*tot_lum); // var / (tot+0.001f);
}

bool ends_with(const std::string& str, const std::string& suffix) {
	return str.size() >= suffix.size() && 0 == str.compare(str.size()-suffix.size(), suffix.size(), suffix);
}

NerfDataset create_empty_nerf_dataset(size_t n_images, int aabb_scale, bool is_hdr) {
	NerfDataset result{};
	result.n_images = n_images;
	result.sharpness_resolution = {128,72};
	result.sharpness_data.enlarge( result.sharpness_resolution.x() * result.sharpness_resolution.y() *  result.n_images );
	result.xforms.resize(n_images);
	result.metadata_normal.resize(n_images);
	result.pixelmemory_normal.resize(n_images);
	result.depthmemory_normal.resize(n_images);
	result.raymemory_normal.resize(n_images);
	result.metadata_albedo.resize(n_images);
	result.pixelmemory_albedo.resize(n_images);
	result.depthmemory_albedo.resize(n_images);
	result.raymemory_albedo.resize(n_images);
	result.scale = NERF_SCALE;
	result.offset = {0.5f, 0.5f, 0.5f};
	result.aabb_scale = aabb_scale;
	result.is_hdr = is_hdr;
	for (size_t i = 0; i < n_images; ++i) {
		result.xforms[i].start = Eigen::Matrix<float, 3, 4>::Identity();
		result.xforms[i].end = Eigen::Matrix<float, 3, 4>::Identity();
	}
	return result;
}

// NerfDataset load_nerf(const std::vector<filesystem::path>& jsonpaths, float sharpen_amount, bool is_downsample) {
NerfDataset load_nerf(const std::vector<filesystem::path>& jsonpaths, float sharpen_amount) {
	tlog::warning()<<"call load_nerf(json,sharpen_amount) in nerf_loader 198";
	if (jsonpaths.empty()) {
		throw std::runtime_error{"Cannot load NeRF data from an empty set of paths."};
	}

	tlog::info() << "Loading NeRF dataset from";

	NerfDataset result{};
	std::ifstream f{jsonpaths.front().str()};
	nlohmann::json transforms = nlohmann::json::parse(f, nullptr, true, true);

	ThreadPool pool;

	struct LoadedImageInfo {
		Eigen::Vector2i res = Eigen::Vector2i::Zero();
		bool image_data_on_gpu = false;
		EImageDataType image_type = EImageDataType::None;
		bool white_transparent = false;
		bool black_transparent = false;
		uint32_t mask_color = 0;
		void *pixels = nullptr;
		uint16_t *depth_pixels = nullptr;
		Ray *rays = nullptr;
		float depth_scale = -1.f;
	};
	std::vector<LoadedImageInfo> images_normal;
	std::vector<LoadedImageInfo> images_albedo;
	LoadedImageInfo info = {};

	if (transforms["camera"].is_array()) {
		throw std::runtime_error{"hdf5 is no longer supported. please use the hdf52nerf.py conversion script"};
	}

	// auto transfer_to_downsample_json = [] (const auto& path, const bool is_downsample) {
	// 	if (!is_downsample) {
	// 		return path.str();
	// 	}
	// 	std::string downsample_path = path.stem().str() + std::string{"_downsample.json"};
	// 	printf("load downsample json: %s\n", downsample_path.c_str());
	// 	return downsample_path;
	// };
	// nerf original format
	std::vector<nlohmann::json> jsons;
	std::transform(
		jsonpaths.begin(), jsonpaths.end(),
		std::back_inserter(jsons), [=] (const auto& path) {
			// return nlohmann::json::parse(std::ifstream{transfer_to_downsample_json(path, is_downsample)}, nullptr, true, true);
			return nlohmann::json::parse(std::ifstream{path.str()}, nullptr, true, true);
		}
	);

	// For dynamic scene: one json for all frame, all images in one frame

	result.n_images = 0;
	fs::path basepath;
	for (size_t i = 0; i < jsons.size(); ++i) {
		auto& json = jsons[i];
		basepath = jsonpaths[i].parent_path();
		if (!json.contains("frames") || !json["frames"].is_array()) {
			tlog::warning() << "  " << jsonpaths[i] << " does not contain any frames. Skipping.";
			continue;
		}
		tlog::info() << "  " << jsonpaths[i];

		result.height = json.value("h",0);
		result.width = json.value("w",0);

		auto& frames = json["frames"];

		float sharpness_discard_threshold = json.value("sharpness_discard_threshold", 0.0f); // Keep all by default

		// std::sort(frames.begin(), frames.end(), [](const auto& frame1, const auto& frame2) {
		// 	return frame1["file_path"] < frame2["file_path"];
		// });

		if (json.contains("n_frames")) {
			size_t cull_idx = std::min(frames.size(), (size_t)json["n_frames"]);
			frames.get_ptr<nlohmann::json::array_t*>()->resize(cull_idx);
		}

		if (frames[0].contains("sharpness")) {
			auto frames_copy = frames;
			frames.clear();

			// Kill blurrier frames than their neighbors
			const int neighborhood_size = 3;
			for (int i = 0; i < (int)frames_copy.size(); ++i) {
				float mean_sharpness = 0.0f;
				int mean_start = std::max(0, i-neighborhood_size);
				int mean_end = std::min(i+neighborhood_size, (int)frames_copy.size()-1);
				for (int j = mean_start; j < mean_end; ++j) {
					mean_sharpness += float(frames_copy[j]["sharpness"]);
				}
				mean_sharpness /= (mean_end - mean_start);

				// Compatibility with Windows paths on Linux. (Breaks linux filenames with "\\" in them, which is acceptable for us.)
				frames_copy[i]["file_path"] = replace_all(frames_copy[i]["file_path"], "\\", "/");

				if ((basepath / fs::path(std::string(frames_copy[i]["file_path"]))).exists() && frames_copy[i]["sharpness"] > sharpness_discard_threshold * mean_sharpness) {
					frames.emplace_back(frames_copy[i]);
				}
			}
		}

		result.n_lights = 1;
		result.n_views = frames.size();
		result.n_images += result.n_views * result.n_lights;
	}


	result.xforms.resize(result.n_views);
	result.metadata_normal.resize(result.n_images);
	result.metadata_albedo.resize(result.n_images);
	images_normal.resize(result.n_images);
	images_albedo.resize(result.n_images);
	result.pixelmemory_normal.resize(result.n_images);
	result.depthmemory_normal.resize(result.n_images);
	result.raymemory_normal.resize(result.n_images);
	result.pixelmemory_albedo.resize(result.n_images);
	result.depthmemory_albedo.resize(result.n_images);
	result.raymemory_albedo.resize(result.n_images);
	

	result.scale = NERF_SCALE;
	result.offset = {0.5f, 0.5f, 0.5f};

	std::vector<std::future<void>> futures;

	size_t image_idx = 0;
	if (result.n_images==0) {
		throw std::invalid_argument{"No training images were found for NeRF training!"};
	}

	auto progress = tlog::progress(result.n_images);

	result.from_mitsuba = false;
	result.from_na = false;
	bool fix_premult = false;
	bool enable_ray_loading = true;
	bool enable_depth_loading = true;
	std::atomic<int> n_loaded{0};
	BoundingBox cam_aabb;

	tlog::success() << "Before loop";
	for (size_t i = 0; i < jsons.size(); ++i) {
		auto& json = jsons[i];

		fs::path basepath = jsonpaths[i].parent_path();
		std::string jp = jsonpaths[i].str();
		tlog::success() << jp;
		auto lastdot=jp.find_last_of('.'); if (lastdot==std::string::npos) lastdot=jp.length();
		auto lastunderscore=jp.find_last_of('_'); if (lastunderscore==std::string::npos) lastunderscore=lastdot; else lastunderscore++;
		std::string part_after_underscore(jp.begin()+lastunderscore,jp.begin()+lastdot);

		if (json.contains("enable_ray_loading")) {
			enable_ray_loading = bool(json["enable_ray_loading"]);
			tlog::info() << "enable_ray_loading=" << enable_ray_loading;
		}
		if (json.contains("enable_depth_loading")) {
			enable_depth_loading = bool(json["enable_depth_loading"]);
			tlog::info() << "enable_depth_loading is " << enable_depth_loading;
		}

		if (json.contains("normal_mts_args")) {
			result.from_mitsuba = true;
		}

		if (json.contains("from_na")) {
			result.from_na = true;
		}

		if (json.contains("fix_premult")) {
			fix_premult = (bool)json["fix_premult"];
		}
		if (result.from_mitsuba) {
			result.scale = 0.66f;
			result.offset = {0.25f * result.scale, 0.25f * result.scale, 0.25f * result.scale};
		}

		if (json.contains("render_aabb")) {
			result.render_aabb.min={float(json["render_aabb"][0][0]),float(json["render_aabb"][0][1]),float(json["render_aabb"][0][2])};
			result.render_aabb.max={float(json["render_aabb"][1][0]),float(json["render_aabb"][1][1]),float(json["render_aabb"][1][2])};
		}

		if (json.contains("sharpen")) {
			sharpen_amount = json["sharpen"];
		}

		if (json.contains("white_transparent")) {
			info.white_transparent = bool(json["white_transparent"]);
		}

		if (json.contains("black_transparent")) {
			info.black_transparent = bool(json["black_transparent"]);
		}

		if (json.contains("scale")) {
			result.scale = json["scale"];
		}
		if (json.contains("importance_sampling")) {
			result.wants_importance_sampling = json["importance_sampling"];
		}

		if (json.contains("n_extra_learnable_dims")) {
			result.n_extra_learnable_dims = json["n_extra_learnable_dims"];
		}

		CameraDistortion camera_distortion = {};
		Vector2f principal_point = Vector2f::Constant(0.5f);
		Vector4f rolling_shutter = Vector4f::Zero();

		if (json.contains("integer_depth_scale")) {
			info.depth_scale = json["integer_depth_scale"];
		}
		// Camera distortion
		{
			if (json.contains("k1")) {
				camera_distortion.params[0] = json["k1"];
				if (camera_distortion.params[0] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("k2")) {
				camera_distortion.params[1] = json["k2"];
				if (camera_distortion.params[1] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("p1")) {
				camera_distortion.params[2] = json["p1"];
				if (camera_distortion.params[2] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("p2")) {
				camera_distortion.params[3] = json["p2"];
				if (camera_distortion.params[3] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("cx")) {
				principal_point.x() = (float)json["cx"] / (float)json["w"];
			}

			if (json.contains("cy")) {
				principal_point.y() = (float)json["cy"] / (float)json["h"];
			}

			if (json.contains("rolling_shutter")) {
				// the rolling shutter is a float3 of [A,B,C] where the time
				// for each pixel is t= A + B * u + C * v
				// where u and v are the pixel coordinates (0-1),
				// and the resulting t is used to interpolate between the start
				// and end transforms for each training xform
				float motionblur_amount = 0.f;
				if (json["rolling_shutter"].size() >= 4) {
					motionblur_amount = float(json["rolling_shutter"][3]);
				}

				rolling_shutter = {float(json["rolling_shutter"][0]), float(json["rolling_shutter"][1]), float(json["rolling_shutter"][2]), motionblur_amount};
			}

			if (json.contains("ftheta_p0")) {
				camera_distortion.params[0] = json["ftheta_p0"];
				camera_distortion.params[1] = json["ftheta_p1"];
				camera_distortion.params[2] = json["ftheta_p2"];
				camera_distortion.params[3] = json["ftheta_p3"];
				camera_distortion.params[4] = json["ftheta_p4"];
				camera_distortion.params[5] = json["w"];
				camera_distortion.params[6] = json["h"];
				camera_distortion.mode = ECameraDistortionMode::FTheta;
			}
		}
		if (json.contains("aabb_scale")) {
			result.aabb_scale = json["aabb_scale"];
		}

		if (json.contains("offset")) {
			result.offset =
				json["offset"].is_array() ?
				Vector3f{float(json["offset"][0]), float(json["offset"][1]), float(json["offset"][2])} :
				Vector3f{float(json["offset"]), float(json["offset"]), float(json["offset"])};
		}

		if (json.contains("aabb")) {
			// map the given aabb of the form [[minx,miny,minz],[maxx,maxy,maxz]] via an isotropic scale and translate to fit in the (0,0,0)-(1,1,1) cube, with the given center at 0.5,0.5,0.5
			const auto& aabb=json["aabb"];
			float length = std::max(0.000001f,std::max(std::max(std::abs(float(aabb[1][0])-float(aabb[0][0])),std::abs(float(aabb[1][1])-float(aabb[0][1]))),std::abs(float(aabb[1][2])-float(aabb[0][2]))));
			result.scale = 1.f/length;
			result.offset = { ((float(aabb[1][0])+float(aabb[0][0]))*0.5f)*-result.scale + 0.5f , ((float(aabb[1][1])+float(aabb[0][1]))*0.5f)*-result.scale + 0.5f,((float(aabb[1][2])+float(aabb[0][2]))*0.5f)*-result.scale + 0.5f};
		}
		if (json.contains("frames") && json["frames"].is_array()) {
			for (int j = 0; j < json["frames"].size(); ++j) {
				auto& frame = json["frames"][j];
				nlohmann::json& jsonmatrix_start = frame.contains("transform_matrix_start") ? frame["transform_matrix_start"] : frame["transform_matrix"];
				nlohmann::json& jsonmatrix_end = frame.contains("transform_matrix_end") ? frame["transform_matrix_end"] : jsonmatrix_start;
				const Vector3f p = Vector3f{float(jsonmatrix_start[0][3]), float(jsonmatrix_start[1][3]), float(jsonmatrix_start[2][3])} * result.scale + result.offset;
				const Vector3f q = Vector3f{float(jsonmatrix_end[0][3]), float(jsonmatrix_end[1][3]), float(jsonmatrix_end[2][3])} * result.scale + result.offset;
				cam_aabb.enlarge(p);
				cam_aabb.enlarge(q);
			}
		}
		if (json.contains("up")) {
			// axes are permuted as for the xforms below
			result.up[0] = float(json["up"][1]);
			result.up[1] = float(json["up"][2]);
			result.up[2] = float(json["up"][0]);
		}
		if (json.contains("envmap") && result.envmap_resolution.isZero()) {
			std::string json_provided_path = json["envmap"];
			fs::path envmap_path = basepath / json_provided_path;
			if (!envmap_path.exists()) {
				throw std::runtime_error{std::string{"Environment map path "} + envmap_path.str() + " does not exist."};
			}

			if (equals_case_insensitive(envmap_path.extension(), "exr")) {
				result.envmap_data = load_exr(envmap_path.str(), result.envmap_resolution.x(), result.envmap_resolution.y());
				result.is_hdr = true;
			} else {
				result.envmap_data = load_stbi(envmap_path.str(), result.envmap_resolution.x(), result.envmap_resolution.y());
			}
		}

		tlog::success() << "Images from MAIN to be loaded !";
		// if (json.contains("frames") && json["frames"].is_array()) pool.parallelForAsync<size_t>(0, json["frames"].size(), [&, basepath, image_idx, info](size_t i) {
		if (json.contains("frames") && json["frames"].is_array()) pool.parallelForAsync<size_t>(0, json["frames"].size(), [&progress, &n_loaded, &result, &images_normal, &images_albedo, &json, basepath, image_idx, info, rolling_shutter, principal_point, camera_distortion, part_after_underscore, fix_premult, enable_depth_loading, enable_ray_loading](size_t i) {
			size_t i_img = i;
			auto& frame = json["frames"][i];

			// LES MATRICES DE R/T POUR CHAQUES VUES

			nlohmann::json& jsonmatrix_start = frame.contains("transform_matrix_start") ? frame["transform_matrix_start"] : frame["transform_matrix"];
			nlohmann::json& jsonmatrix_end =   frame.contains("transform_matrix_end") ? frame["transform_matrix_end"] : jsonmatrix_start;

			for (int m = 0; m < 3; ++m) {
				for (int n = 0; n < 4; ++n) {
					result.xforms[i_img].start(m, n) = float(jsonmatrix_start[m][n]);
					result.xforms[i_img].end(m, n) = float(jsonmatrix_end[m][n]);
				}
			}

			result.xforms[i_img].start = result.nerf_matrix_to_ngp(result.xforms[i_img].start);
			result.xforms[i_img].end = result.nerf_matrix_to_ngp(result.xforms[i_img].end);


			if (json.contains("n2w")) {
				for (int m = 0; m < 3; ++m) {
					result.n2w_t(m) = float(json["n2w"][m][3]);
				}
				result.n2w_s = float(json["n2w"][0][0]);
			}	

			//////////////////////////////////////////////////////////////////////////////////////
			/// NORMAL IMAGE

			LoadedImageInfo& dst_normal = images_normal[i_img];
			dst_normal = info; // copy defaults

			std::string json_provided_path_normal(frame["normal_path"]);
			if (json_provided_path_normal == "") {
				char buf[256];
				snprintf(buf, 256, "%s_%03d/rgba.png", part_after_underscore.c_str(), (int)i);
				json_provided_path_normal = buf;
			}
			fs::path path_normal = basepath / json_provided_path_normal;

			if (path_normal.extension() == "") {
				path_normal = path_normal.with_extension("png");
				if (!path_normal.exists()) {
					path_normal = path_normal.with_extension("exr");
				}
				if (!path_normal.exists()) {
					throw std::runtime_error{ "Could not find image file: " + path_normal.str()};
				}
			}

			std::string img_path_normal = path_normal.str();
			replace(img_path_normal.begin(),img_path_normal.end(),'\\','/');

			int comp_normal = 0;
			
			dst_normal.image_data_on_gpu = false;
			// uint8_t* img = stbi_load(path.str().c_str(), &dst.res.x(), &dst.res.y(), &comp, 4);
			uint16_t* img_normal = stbi_load_16(img_path_normal.c_str(), &dst_normal.res.x(), &dst_normal.res.y(), &comp_normal, 4);

			dst_normal.pixels = img_normal;
			dst_normal.image_type = EImageDataType::Byte;

			if (!dst_normal.pixels) {
				// throw std::runtime_error{ "image not found: " + path.str() };
				throw std::runtime_error{ "image not found: " + img_path_normal };
			}

			/////////////////////////////////////////////////////////////////////////////////////////
			/// ALBEDO IMAGE

			LoadedImageInfo& dst_albedo = images_albedo[i_img];
			dst_albedo = info; // copy defaults

			std::string json_provided_path_albedo(frame["albedo_path"]);
			if (json_provided_path_albedo == "") {
				char buf[256];
				snprintf(buf, 256, "%s_%03d/rgba.png", part_after_underscore.c_str(), (int)i);
				json_provided_path_albedo = buf;
			}
			fs::path path_albedo = basepath / json_provided_path_albedo;

			if (path_albedo.extension() == "") {
				path_albedo = path_albedo.with_extension("png");
				if (!path_albedo.exists()) {
					path_albedo = path_albedo.with_extension("exr");
				}
				if (!path_albedo.exists()) {
					throw std::runtime_error{ "Could not find image file: " + path_albedo.str()};
				}
			}

			std::string img_path_albedo = path_albedo.str();
			replace(img_path_albedo.begin(),img_path_albedo.end(),'\\','/');

			int comp_albedo = 0;
			
			dst_albedo.image_data_on_gpu = false;
			// uint8_t* img = stbi_load(path.str().c_str(), &dst.res.x(), &dst.res.y(), &comp, 4);
			uint16_t* img_albedo = stbi_load_16(img_path_albedo.c_str(), &dst_albedo.res.x(), &dst_albedo.res.y(), &comp_albedo, 4);

			dst_albedo.pixels = img_albedo;
			dst_albedo.image_type = EImageDataType::Byte;

			if (!dst_albedo.pixels) {
				// throw std::runtime_error{ "image not found: " + path.str() };
				throw std::runtime_error{ "image not found: " + img_path_albedo };
			}
 
			///////////////////////////////////////////////////////////////////////////////////////////////////////

			auto read_focal_length = [&](int resolution, const std::string& axis) {
				if (frame.contains(axis + "_fov")) {
					return fov_to_focal_length(resolution, (float)frame[axis + "_fov"]);
				} else if (json.contains("fl_"s + axis)) {
					return (float)json["fl_"s + axis];
				} else if (json.contains("camera_angle_"s + axis)) {
					return fov_to_focal_length(resolution, (float)json["camera_angle_"s + axis] * 180 / PI());
				} else {
					return 0.0f;
				}
			};


			
			const auto& intrinsic = frame["intrinsic_matrix"];
			result.metadata_normal[i_img].focal_length.x() = float(intrinsic[0][0]);
			result.metadata_normal[i_img].focal_length.y() = float(intrinsic[1][1]);
			result.metadata_normal[i_img].s0 = float(intrinsic[0][1]);
			result.metadata_normal[i_img].principal_point.x() = float(intrinsic[0][2])/(float)json["w"];
			result.metadata_normal[i_img].principal_point.y() = float(intrinsic[1][2])/(float)json["h"];
			result.metadata_albedo[i_img].focal_length.x() = float(intrinsic[0][0]);
			result.metadata_albedo[i_img].focal_length.y() = float(intrinsic[1][1]);
			result.metadata_albedo[i_img].s0 = float(intrinsic[0][1]);
			result.metadata_albedo[i_img].principal_point.x() = float(intrinsic[0][2])/(float)json["w"];
			result.metadata_albedo[i_img].principal_point.y() = float(intrinsic[1][2])/(float)json["h"];	
				

			result.metadata_normal[i_img].rolling_shutter = rolling_shutter;
			result.metadata_normal[i_img].camera_distortion = camera_distortion;
			result.metadata_albedo[i_img].rolling_shutter = rolling_shutter;
			result.metadata_albedo[i_img].camera_distortion = camera_distortion;


			progress.update(++n_loaded);
		}, futures);
		
		if (json.contains("frames")) {
			image_idx += json["frames"].size();
		}

		}

	waitAll(futures);

	tlog::success() << "Loaded " << images_albedo.size() << " images after " << tlog::durationToString(progress.duration());
	tlog::info() << "  cam_aabb=" << cam_aabb;


	result.sharpness_resolution = { 128, 72 };
	result.sharpness_data.enlarge( result.sharpness_resolution.x() * result.sharpness_resolution.y() *  result.n_images );

	////////////////////////////////////////////////////////////////////////////////////
	/// NORMAL IMAGES
	// copy / convert images to the GPU
	for (uint32_t i = 0; i < result.n_images; ++i) {
		const LoadedImageInfo& m_normal = images_normal[i];
		result.set_training_image_normal(i, m_normal.res, m_normal.pixels, m_normal.depth_pixels, m_normal.depth_scale * result.scale, m_normal.image_data_on_gpu, m_normal.image_type, 
		EDepthDataType::UShort, sharpen_amount, m_normal.white_transparent, m_normal.black_transparent, m_normal.mask_color, m_normal.rays);
		CUDA_CHECK_THROW(hipDeviceSynchronize());
	}
	CUDA_CHECK_THROW(hipDeviceSynchronize());


	// free memory
	for (uint32_t i = 0; i < result.n_images; ++i) {
		if (images_normal[i].image_data_on_gpu) {
			CUDA_CHECK_THROW(hipFree(images_normal[i].pixels));
		} else {
			free(images_normal[i].pixels);
		}
		free(images_normal[i].rays);
		free(images_normal[i].depth_pixels);
	}

	////////////////////////////////////////////////////////////////////////////////////
	/// ALBEDO IMAGES
	// copy / convert images to the GPU
	for (uint32_t i = 0; i < result.n_images; ++i) {
		const LoadedImageInfo& m_albedo = images_albedo[i];
		result.set_training_image_albedo(i, m_albedo.res, m_albedo.pixels, m_albedo.depth_pixels, m_albedo.depth_scale * result.scale, m_albedo.image_data_on_gpu, m_albedo.image_type, 
		EDepthDataType::UShort, sharpen_amount, m_albedo.white_transparent, m_albedo.black_transparent, m_albedo.mask_color, m_albedo.rays);
		CUDA_CHECK_THROW(hipDeviceSynchronize());
	}
	CUDA_CHECK_THROW(hipDeviceSynchronize());


	// free memory
	for (uint32_t i = 0; i < result.n_images; ++i) {
		if (images_albedo[i].image_data_on_gpu) {
			CUDA_CHECK_THROW(hipFree(images_albedo[i].pixels));
		} else {
			free(images_albedo[i].pixels);
		}
		free(images_albedo[i].rays);
		free(images_albedo[i].depth_pixels);
	}


	return result;
}

void NerfDataset::set_training_image_normal(int frame_idx, const Eigen::Vector2i& image_resolution, const void* pixels, const void* depth_pixels, float depth_scale, bool image_data_on_gpu, EImageDataType image_type, EDepthDataType depth_type, float sharpen_amount, bool white_transparent, bool black_transparent, uint32_t mask_color, const Ray *rays) {
	if (frame_idx < 0 || frame_idx >= n_images) {
		throw std::runtime_error{"NerfDataset::set_training_image: invalid frame index"};
	}
	size_t n_pixels = image_resolution.prod();
	size_t img_size = n_pixels * 4; // 4 channels
	size_t image_type_stride = image_type_size(image_type);
	// copy to gpu if we need to do a conversion
	GPUMemory<uint16_t> images_data_gpu_tmp;
	GPUMemory<uint8_t> depth_tmp;
	if (!image_data_on_gpu && image_type == EImageDataType::Byte) {
		images_data_gpu_tmp.resize(img_size * image_type_stride);
		images_data_gpu_tmp.copy_from_host((uint16_t*)pixels);
		pixels = images_data_gpu_tmp.data();

		if (depth_pixels) {
			depth_tmp.resize(n_pixels * depth_type_size(depth_type));
			depth_tmp.copy_from_host((uint8_t*)depth_pixels);
			depth_pixels = depth_tmp.data();
		}

		image_data_on_gpu = true;
	}

	// copy or convert the pixels
	pixelmemory_normal[frame_idx].resize(img_size * image_type_size(image_type));
	void* dst = pixelmemory_normal[frame_idx].data();

	switch (image_type) {
		default: throw std::runtime_error{"unknown image type in set_training_image"};
		case EImageDataType::Byte: linear_kernel(convert_rgba64, 0, nullptr, n_pixels, (uint16_t*)pixels, (uint16_t*)dst, white_transparent, black_transparent, mask_color); break;
		case EImageDataType::Half: // fallthrough is intended
		case EImageDataType::Float: CUDA_CHECK_THROW(hipMemcpy(dst, pixels, img_size * image_type_size(image_type), image_data_on_gpu ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice)); break;
	}

	// copy over depths if provided
	if (depth_scale >= 0.f) {
		depthmemory_normal[frame_idx].resize(img_size);
		float* depth_dst = depthmemory_normal[frame_idx].data();

		if (depth_pixels && !image_data_on_gpu) {
			depth_tmp.resize(n_pixels * depth_type_size(depth_type));
			depth_tmp.copy_from_host((uint8_t*)depth_pixels);
			depth_pixels = depth_tmp.data();
		}

		switch (depth_type) {
			default: throw std::runtime_error{"unknown depth type in set_training_image"};
			case EDepthDataType::UShort: linear_kernel(copy_depth<uint16_t>, 0, nullptr, n_pixels, depth_dst, (const uint16_t*)depth_pixels, depth_scale); break;
			case EDepthDataType::Float: linear_kernel(copy_depth<float>, 0, nullptr, n_pixels, depth_dst, (const float*)depth_pixels, depth_scale); break;
		}
	} else {
		depthmemory_normal[frame_idx].free_memory();
	}

	// apply requested sharpening
	if (sharpen_amount > 0.f) {
		if (image_type == EImageDataType::Byte) {
			tcnn::GPUMemory<uint16_t> images_data_half(img_size * sizeof(__half));
			linear_kernel(from_rgba64<__half>, 0, nullptr, n_pixels, (uint16_t*)pixels, (__half*)images_data_half.data(), white_transparent, black_transparent, mask_color);
			pixelmemory_normal[frame_idx] = std::move(images_data_half);
			dst = pixelmemory_normal[frame_idx].data();
			image_type = EImageDataType::Half;
		}

		assert(image_type == EImageDataType::Half || image_type == EImageDataType::Float);

		tcnn::GPUMemory<uint16_t> images_data_sharpened(img_size * image_type_size(image_type));

		float center_w = 4.f + 1.f / sharpen_amount; // center_w ranges from 5 (strong sharpening) to infinite (no sharpening)
		if (image_type == EImageDataType::Half) {
			linear_kernel(sharpen<__half>, 0, nullptr, n_pixels, image_resolution.x(), (__half*)dst, (__half*)images_data_sharpened.data(), center_w, 1.f / (center_w - 4.f));
		} else {
			linear_kernel(sharpen<float>, 0, nullptr, n_pixels, image_resolution.x(), (float*)dst, (float*)images_data_sharpened.data(), center_w, 1.f / (center_w - 4.f));
		}

		pixelmemory_normal[frame_idx] = std::move(images_data_sharpened);
		dst = pixelmemory_normal[frame_idx].data();
	}

	if (sharpness_data.size()>0) {
		// compute overall sharpness
		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)sharpness_resolution.x(), threads.x), div_round_up((uint32_t)sharpness_resolution.y(), threads.y), 1 };
		sharpness_data.enlarge(sharpness_resolution.x() * sharpness_resolution.y());
		compute_sharpness<<<blocks, threads, 0, nullptr>>>(sharpness_resolution, image_resolution, 1, dst, image_type, sharpness_data.data() + sharpness_resolution.x() * sharpness_resolution.y() * (size_t)frame_idx);
	}


	metadata_normal[frame_idx].pixels = pixelmemory_normal[frame_idx].data();
	metadata_normal[frame_idx].depth = depthmemory_normal[frame_idx].data();
	metadata_normal[frame_idx].resolution = image_resolution;
	metadata_normal[frame_idx].image_data_type = image_type;
	if (rays) {
		raymemory_normal[frame_idx].resize(n_pixels);
		CUDA_CHECK_THROW(hipMemcpy(raymemory_normal[frame_idx].data(), rays, n_pixels * sizeof(Ray), hipMemcpyHostToDevice));
	} else {
		raymemory_normal[frame_idx].free_memory();
	}
	metadata_normal[frame_idx].rays = raymemory_normal[frame_idx].data();

	
}

void NerfDataset::set_training_image_albedo(int frame_idx, const Eigen::Vector2i& image_resolution, const void* pixels, const void* depth_pixels, float depth_scale, bool image_data_on_gpu, EImageDataType image_type, EDepthDataType depth_type, float sharpen_amount, bool white_transparent, bool black_transparent, uint32_t mask_color, const Ray *rays) {
	if (frame_idx < 0 || frame_idx >= n_images) {
		throw std::runtime_error{"NerfDataset::set_training_image: invalid frame index"};
	}
	size_t n_pixels = image_resolution.prod();
	size_t img_size = n_pixels * 4; // 4 channels
	size_t image_type_stride = image_type_size(image_type);
	// copy to gpu if we need to do a conversion
	GPUMemory<uint16_t> images_data_gpu_tmp;
	GPUMemory<uint8_t> depth_tmp;
	if (!image_data_on_gpu && image_type == EImageDataType::Byte) {
		images_data_gpu_tmp.resize(img_size * image_type_stride);
		images_data_gpu_tmp.copy_from_host((uint16_t*)pixels);
		pixels = images_data_gpu_tmp.data();

		if (depth_pixels) {
			depth_tmp.resize(n_pixels * depth_type_size(depth_type));
			depth_tmp.copy_from_host((uint8_t*)depth_pixels);
			depth_pixels = depth_tmp.data();
		}

		image_data_on_gpu = true;
	}

	// copy or convert the pixels
	pixelmemory_albedo[frame_idx].resize(img_size * image_type_size(image_type));
	void* dst = pixelmemory_albedo[frame_idx].data();

	switch (image_type) {
		default: throw std::runtime_error{"unknown image type in set_training_image"};
		case EImageDataType::Byte: linear_kernel(convert_rgba64, 0, nullptr, n_pixels, (uint16_t*)pixels, (uint16_t*)dst, white_transparent, black_transparent, mask_color); break;
		case EImageDataType::Half: // fallthrough is intended
		case EImageDataType::Float: CUDA_CHECK_THROW(hipMemcpy(dst, pixels, img_size * image_type_size(image_type), image_data_on_gpu ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice)); break;
	}

	// copy over depths if provided
	if (depth_scale >= 0.f) {
		depthmemory_albedo[frame_idx].resize(img_size);
		float* depth_dst = depthmemory_albedo[frame_idx].data();

		if (depth_pixels && !image_data_on_gpu) {
			depth_tmp.resize(n_pixels * depth_type_size(depth_type));
			depth_tmp.copy_from_host((uint8_t*)depth_pixels);
			depth_pixels = depth_tmp.data();
		}

		switch (depth_type) {
			default: throw std::runtime_error{"unknown depth type in set_training_image"};
			case EDepthDataType::UShort: linear_kernel(copy_depth<uint16_t>, 0, nullptr, n_pixels, depth_dst, (const uint16_t*)depth_pixels, depth_scale); break;
			case EDepthDataType::Float: linear_kernel(copy_depth<float>, 0, nullptr, n_pixels, depth_dst, (const float*)depth_pixels, depth_scale); break;
		}
	} else {
		depthmemory_albedo[frame_idx].free_memory();
	}

	// apply requested sharpening
	if (sharpen_amount > 0.f) {
		if (image_type == EImageDataType::Byte) {
			tcnn::GPUMemory<uint16_t> images_data_half(img_size * sizeof(__half));
			linear_kernel(from_rgba64<__half>, 0, nullptr, n_pixels, (uint16_t*)pixels, (__half*)images_data_half.data(), white_transparent, black_transparent, mask_color);
			pixelmemory_albedo[frame_idx] = std::move(images_data_half);
			dst = pixelmemory_albedo[frame_idx].data();
			image_type = EImageDataType::Half;
		}

		assert(image_type == EImageDataType::Half || image_type == EImageDataType::Float);

		tcnn::GPUMemory<uint16_t> images_data_sharpened(img_size * image_type_size(image_type));

		float center_w = 4.f + 1.f / sharpen_amount; // center_w ranges from 5 (strong sharpening) to infinite (no sharpening)
		if (image_type == EImageDataType::Half) {
			linear_kernel(sharpen<__half>, 0, nullptr, n_pixels, image_resolution.x(), (__half*)dst, (__half*)images_data_sharpened.data(), center_w, 1.f / (center_w - 4.f));
		} else {
			linear_kernel(sharpen<float>, 0, nullptr, n_pixels, image_resolution.x(), (float*)dst, (float*)images_data_sharpened.data(), center_w, 1.f / (center_w - 4.f));
		}

		pixelmemory_albedo[frame_idx] = std::move(images_data_sharpened);
		dst = pixelmemory_albedo[frame_idx].data();
	}

	if (sharpness_data.size()>0) {
		// compute overall sharpness
		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)sharpness_resolution.x(), threads.x), div_round_up((uint32_t)sharpness_resolution.y(), threads.y), 1 };
		sharpness_data.enlarge(sharpness_resolution.x() * sharpness_resolution.y());
		compute_sharpness<<<blocks, threads, 0, nullptr>>>(sharpness_resolution, image_resolution, 1, dst, image_type, sharpness_data.data() + sharpness_resolution.x() * sharpness_resolution.y() * (size_t)frame_idx);
	}


	metadata_albedo[frame_idx].pixels = pixelmemory_albedo[frame_idx].data();
	metadata_albedo[frame_idx].depth = depthmemory_albedo[frame_idx].data();
	metadata_albedo[frame_idx].resolution = image_resolution;
	metadata_albedo[frame_idx].image_data_type = image_type;
	if (rays) {
		raymemory_albedo[frame_idx].resize(n_pixels);
		CUDA_CHECK_THROW(hipMemcpy(raymemory_albedo[frame_idx].data(), rays, n_pixels * sizeof(Ray), hipMemcpyHostToDevice));
	} else {
		raymemory_albedo[frame_idx].free_memory();
	}
	metadata_albedo[frame_idx].rays = raymemory_albedo[frame_idx].data();

	
}


NGP_NAMESPACE_END
